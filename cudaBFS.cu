#include "hip/hip_runtime.h"
#include "driverBFS.c"
#include "cudaBFS.h"

__global__ void kernel_set_frontier(gpudata data, csrdata csrg) {

    const int WARPS_PER_BLOCK = blockDim.x/WARP;
    int i, j, warp_id, increment;
    UL V, s, e, *node;

    *(data.redo) = 0;
    warp_id = blockIdx.x*WARPS_PER_BLOCK + threadIdx.x/WARP;
	increment = (gridDim.x*blockDim.x)/WARP;

	for(i = warp_id; i < csrg.nv; i+= increment) {
	    if (data.queue[i]) {

            data.queue[i] = 0;
            s = csrg.offsets[i];
            e = csrg.offsets[i+1] - s;

            node = &csrg.rows[s];

            for (j = threadIdx.x%WARP; j < e; j += WARP) {
                V = node[j];
                data.frontier[V] = 1;
            }
        }
    }
}

__global__ void kernel_compute_distance(gpudata data) {
    UL prev_level, dist;

    int tid = (blockIdx.x*blockDim.x)+threadIdx.x;

    while (tid < data.vertex) {
        if (data.frontier[tid]) {
            *(data.redo) = 1;
            data.frontier[tid] = 0;
            prev_level = data.dist[tid];
			dist = data.level + 1;
            data.dist[tid] = (dist < prev_level) ? dist : prev_level;
            data.queue[tid] = (prev_level == ULONG_MAX);
        }
        tid += gridDim.x*blockDim.x;
    }
}

UL *do_bfs_cuda(UL source, csrdata *csrgraph, csrdata *csrgraph_gpu, double *cudatime)
{
    UL U, V, s, e, i, j;

    // Creo le strutture per i timer
    hipEvent_t exec_start, exec_stop, alloc_copy_start, alloc_copy_stop;
    double alloc_copy_time = 0.0, bfs_time = 0.0;
    char redo = 1;

    // Dati per la gpu
    gpudata host;
    gpudata dev;

    // Variabili per ottimizzare la bfs
    int gpu, num_threads, num_blocks;
    hipDeviceProp_t gpu_prop;

    // Leggo le proprietà del device per ottimizzare la bfs
    hipGetDevice(&gpu);
    hipGetDeviceProperties(&gpu_prop, gpu);
    num_threads = gpu_prop.maxThreadsPerBlock;
    num_blocks = csrgraph->nv/num_threads;
	if((csrgraph->nv % num_threads) > 0) num_blocks++;
    printf("\nNumber of threads: %d,\tNumber of blocks: %d\n", num_threads, num_blocks);

    // Inizializzo i dati
    host.level = 0;
    host.queue = (char *) Malloc(csrgraph->nv);
    host.dist = (UL *) Malloc(csrgraph->nv*sizeof(UL));
    host.vertex = csrgraph->nv;
    memset(host.queue, 0, csrgraph->nv);
    for (i = 0; i < csrgraph->nv; i++) host.dist[i] = ULONG_MAX;

    // La prima iterazione la faccio seriale
    host.dist[source] = 0;

    // dequeue U
    U = source;
    // Search all neighbors of U
    s = csrgraph->offsets[U]; e = csrgraph->offsets[U+1];
    for (j = s; j < e; j++) {
        V = csrgraph->rows[j];
        // If V is not visited enqueue it and set its distance
        host.queue[V] = 1;
        host.dist[V]  = host.level + 1;
    }
    host.level += 1;
    dev.level = host.level;

    // Inizio ad allocare memoria e copiare i dati sulla gpu
    START_CUDA_TIMER(&alloc_copy_start, &alloc_copy_stop);
    copy_data_on_gpu(&host, &dev);
    alloc_copy_time = STOP_CUDA_TIMER(&alloc_copy_start, &alloc_copy_stop);
    printf("\nTime spent for allocation and copy: %.5f\n", alloc_copy_time);

    // Faccio partire il kernel
    START_CUDA_TIMER(&exec_start, &exec_stop);
    while(redo) {
        // lancio il kernel
        kernel_set_frontier<<<num_blocks, num_threads>>>(dev, *csrgraph_gpu);
        kernel_compute_distance<<<num_blocks, num_threads>>>(dev);
        dev.level += 1;
        HANDLE_ERROR(hipMemcpy(&redo, (&dev)->redo, sizeof(char), hipMemcpyDeviceToHost));
    }
    bfs_time = STOP_CUDA_TIMER(&exec_start, &exec_stop);
    printf("Time spent for cuda bfs: %.5f\n", bfs_time);

    copy_data_on_host(&host, &dev);
    free_gpu_mem(&dev, csrgraph_gpu);

    *cudatime = alloc_copy_time + bfs_time;
    return host.dist;
}

UL *traverse_parallel(UL *edges, UL nedges, UL nvertices, UL root, int randsource, int seed)
{
    csrdata csrgraph, csrgraph_gpu;     // csr data structure to represent the graph
	FILE *fout;
	UL i;
	UL *dist;             // array of distances from the source

	// Vars for timing
	struct timeval begin, end;
	double cudatime = 0.0, csrtime;
	int timer = 1;

	csrgraph.offsets = NULL;
	csrgraph.rows    = NULL;
	csrgraph.deg     = NULL;

	// Build the CSR data structure
	START_TIMER(begin)
	csrgraph.offsets = (UL *)Malloc((nvertices+1)*sizeof(UL));
	csrgraph.rows    = (UL *)Malloc(nedges       *sizeof(UL));
	csrgraph.deg     = (UL *)Malloc(nvertices    *sizeof(UL));

	build_csr(edges, nedges, nvertices, &csrgraph);
    copy_csr_on_gpu(&csrgraph, &csrgraph_gpu);
	END_TIMER(end);
	ELAPSED_TIME(csrtime, begin, end)
	if (randsource) {
		root = random_source(&csrgraph, seed);
		fprintf(stdout, "Random source vertex %lu\n", root);
	}

    dist = do_bfs_cuda(root, &csrgraph, &csrgraph_gpu, &cudatime);

	// Print distance array to file
	fout = Fopen(DISTANCE_OUT_FILE, "w+");
	for (i = 0; i < csrgraph.nv; i++) fprintf(fout, "%lu %lu\n", i, dist[i]);
	fclose(fout);

	// Timing output
    fprintf(stdout, "\n");
	fprintf(stdout, "Cuda build csr and copy time = \t%.5f\n", csrtime);
	fprintf(stdout, "Cuda alloc data and bfs time = \t%.5f\n", cudatime);
	fprintf(stdout, "\n");

	if(csrgraph.offsets) free(csrgraph.offsets);
	if(csrgraph.rows)    free(csrgraph.rows);

	return dist;
}
