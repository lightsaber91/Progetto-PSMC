#include "hip/hip_runtime.h"
#include "driverBFS.c"
#include "cudaBFS.h"

__global__ void kernel_set_frontier(gpudata data, csrdata csrg) {

    int warp_size = data.warp_size;
    int warps_block = blockDim.x / warp_size;
    int i, j, warp_id, increment;
    UL V, s, e, *node;

    *(data.redo) = 0;
    warp_id = blockIdx.x * warps_block + threadIdx.x / warp_size;
    increment = (gridDim.x * blockDim.x)/warp_size;

    for(i = warp_id; i < csrg.nv; i+= increment) {
        if (data.queue[i]) {

            data.queue[i] = 0;
            s = csrg.offsets[i];
            e = csrg.offsets[i+1] - s;

            node = &csrg.rows[s];

            for (j = threadIdx.x % warp_size; j < e; j += warp_size) {
                V = node[j];
                data.frontier[V] = 1;
            }
        }
    }
}

__global__ void kernel_compute_distance(gpudata data) {
    UL prev_level, dist;

    int tid = (blockIdx.x*blockDim.x)+threadIdx.x;

    while (tid < data.vertex) {
        if (data.frontier[tid]) {
            *(data.redo) = 1;
            data.frontier[tid] = 0;
            prev_level = data.dist[tid];
            dist = data.level + 1;
            data.dist[tid] = (dist < prev_level) ? dist : prev_level;
            data.queue[tid] = (prev_level == ULONG_MAX);
        }
        tid += gridDim.x*blockDim.x;
    }
}

UL *do_bfs_cuda(UL source, csrdata *csrgraph, csrdata *csrgraph_gpu, double *cudatime, int thread, int counter)
{
    int num_threads, num_blocks, i;

    // Creo le strutture per i timer
    hipEvent_t exec_start, exec_stop, alloc_copy_start, alloc_copy_stop;
    double alloc_copy_time = 0.0, bfs_time = 0.0;
    char redo = 1;

    // Dati per la gpu
    gpudata host;
    gpudata dev;

    // Leggo le proprietà del device per ottimizzare la bfs
    set_threads_and_blocks(&num_threads, &num_blocks, &(dev.warp_size), csrgraph->nv, thread);
    if(counter == 0) printf("\nNumber of threads: %d,\tNumber of blocks: %d\n", num_threads, num_blocks);

    // Inizializzo i dati
    host.level = 0;
    host.queue = (char *) Malloc(csrgraph->nv);
    host.dist = (UL *) Malloc(csrgraph->nv*sizeof(UL));
    host.vertex = csrgraph->nv;
    memset(host.queue, 0, csrgraph->nv);
    for (i = 0; i < csrgraph->nv; i++) host.dist[i] = ULONG_MAX;

    host.dist[source] = 0;
    host.queue[source] = 1;
    dev.level = host.level;

    // Inizio ad allocare memoria e copiare i dati sulla gpu
    START_CUDA_TIMER(&alloc_copy_start, &alloc_copy_stop);
    copy_data_on_gpu(&host, &dev);
    alloc_copy_time = STOP_CUDA_TIMER(&alloc_copy_start, &alloc_copy_stop);
//    printf("\nTime spent for allocation and copy: %.5f\n", alloc_copy_time);

    // Faccio partire il kernel
    START_CUDA_TIMER(&exec_start, &exec_stop);
    while(redo) {
        // lancio il kernel
        kernel_set_frontier<<<num_blocks, num_threads>>>(dev, *csrgraph_gpu);
        kernel_compute_distance<<<num_blocks, num_threads>>>(dev);
        dev.level += 1;
        HANDLE_ERROR(hipMemcpy(&redo, (&dev)->redo, sizeof(char), hipMemcpyDeviceToHost));
    }
    bfs_time = STOP_CUDA_TIMER(&exec_start, &exec_stop);
//    printf("Time spent for cuda bfs: %.5f\n", bfs_time);

    copy_data_on_host(&host, &dev);
    free_gpu_mem(&dev);
    free(host.queue);

    *cudatime = bfs_time;
    return host.dist;
}

UL *traverse_parallel(UL *edges, UL nedges, UL nvertices, UL root, int randsource, int seed, int thread)
{
    csrdata csrgraph, csrgraph_gpu;     // csr data structure to represent the graph
    FILE *fout;
    UL i;
    UL *dist;             // array of distances from the source

    // Vars for timing
    struct timeval begin, end;
    double cudatime = 0.0, csrtime, tottime = 0.0;
    int timer = 1;

    csrgraph.offsets = NULL;
    csrgraph.rows    = NULL;
    csrgraph.deg     = NULL;

    // Build the CSR data structure
    START_TIMER(begin)
    csrgraph.offsets = (UL *)Malloc((nvertices+1)*sizeof(UL));
    csrgraph.rows    = (UL *)Malloc(nedges       *sizeof(UL));
    csrgraph.deg     = (UL *)Malloc(nvertices    *sizeof(UL));

    build_csr(edges, nedges, nvertices, &csrgraph);
    copy_csr_on_gpu(&csrgraph, &csrgraph_gpu);
    END_TIMER(end);
    ELAPSED_TIME(csrtime, begin, end)

    if (randsource) {
        root = random_source(&csrgraph, seed);
        fprintf(stdout, "Random source vertex %lu\n", root);
    }
//    while( thread <= 1024) {
        for(i = 0; i < 10; i++) {
          printf("%d", i);
            dist = do_bfs_cuda(root, &csrgraph, &csrgraph_gpu, &cudatime, thread, i);
            sleep(5);
            tottime += cudatime;
            free(dist);
        }
        cudatime = tottime / 10;
    // Print distance array to file
//    fout = Fopen(DISTANCE_OUT_FILE, "w+");
//    for (i = 0; i < csrgraph.nv; i++) fprintf(fout, "%lu %lu\n", i, dist[i]);
//    fclose(fout);

    // Timing output
        fprintf(stdout, "\n");
        fprintf(stdout, "Cuda build csr and copy time = \t%.5f\n", csrtime);
        fprintf(stdout, "Cuda alloc data and bfs time = \t%.5f\n", cudatime);
        fprintf(stdout, "\n");
//        if(thread == -1) thread = 32;
//        else thread *= 2;
//    }
    free_csrgraph_dev(&csrgraph_gpu);

    if(csrgraph.offsets) free(csrgraph.offsets);
    if(csrgraph.rows)    free(csrgraph.rows);

    return dist;
}
